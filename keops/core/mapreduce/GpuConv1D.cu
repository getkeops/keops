#include "hip/hip_runtime.h"
#pragma once

#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "core/pack/Pack.h"
#include "core/pack/GetInds.h"
#include "core/pack/GetDims.h"
#include "core/utils/CudaErrorCheck.cu"
#include "core/utils/CudaSizes.h"

namespace keops {

template<typename TYPE, class FUN>
__global__ void GpuConv1DOnDevice(FUN fun, int nx, int ny, TYPE **px, TYPE **py, TYPE **pp) {

  // get the index of the current thread
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // declare shared mem
  extern __shared__ TYPE
  yj[];

  // get templated dimensions :
  typedef typename FUN::DIMSX DIMSX;  // DIMSX is a "vector" of templates giving dimensions of xi variables
  typedef typename FUN::DIMSY DIMSY;  // DIMSY is a "vector" of templates giving dimensions of yj variables
  typedef typename FUN::DIMSP DIMSP;  // DIMSP is a "vector" of templates giving dimensions of parameters variables
  const int DIMX = DIMSX::SUM;        // DIMX  is sum of dimensions for xi variables
  const int DIMY = DIMSY::SUM;        // DIMY  is sum of dimensions for yj variables
  const int DIMP = DIMSP::SUM;        // DIMP  is sum of dimensions for parameters variables
  const int DIMOUT = FUN::DIM; // dimension of output variable
  const int DIMRED = FUN::DIMRED; // dimension of reduction operation
  const int DIMFOUT = DIMSX::FIRST;     // DIMFOUT is dimension of output variable of inner function

  // load parameter(s)
  TYPE param_loc[DIMP < 1 ? 1 : DIMP];
  load<DIMSP>(0, param_loc, pp); // load parameters variables from global memory to local thread memory

  // get the value of variable (index with i)
  TYPE xi[DIMX < 1 ? 1 : DIMX], tmp[DIMRED], tmp_block[DIMRED]; 
  if (i < nx) {
    typename FUN::template InitializeReduction<TYPE>()(tmp); // tmp = 0
    typename FUN::template InitializeReduction<TYPE>()(tmp_block); // tmp = 0
    load<typename DIMSX::NEXT>(i, xi + DIMFOUT, px + 1); // load xi variables from global memory to local thread memory
  }

  for (int jstart = 0, tile = 0; jstart < ny; jstart += blockDim.x, tile++) {

    // get the current column
    int j = tile * blockDim.x + threadIdx.x;

    if (j < ny) { // we load yj from device global memory only if j<ny
      load<DIMSY>(j, yj + threadIdx.x * DIMY, py); // load yj variables from global memory to shared memory
    }
    __syncthreads();

    if (i < nx) { // we compute x1i only if needed
      TYPE * yjrel = yj; // Loop on the columns of the current block.
      for (int jrel = 0; (jrel < blockDim.x) && (jrel < ny - jstart); jrel++, yjrel += DIMY) {
        call<DIMSX, DIMSY, DIMSP>(fun,
                                  xi,
                                  yjrel,
                                  param_loc); // Call the function, which outputs results in xi[0:DIMX1]
        #pragma unroll
	for (int k=0; k<DIMRED; k++)
        {
		TYPE tmp1 = xi[k] - tmp_block[k];
		TYPE tmp2 = tmp[k] + tmp1;
		tmp_block[k] = (tmp2 - tmp[k]) - tmp1;
		tmp[k] = tmp2;
	}
      }
    }
    __syncthreads();
  }
  if (i < nx) {
    typename FUN::template FinalizeOutput<TYPE>()(tmp, px[0] + i * DIMOUT, px, i);
  }

}

struct GpuConv1D_FromHost {

  template<typename TYPE, class FUN>
  static int Eval_(FUN fun, int nx, int ny, TYPE **px_h, TYPE **py_h, TYPE **pp_h) {

    typedef typename FUN::DIMSX DIMSX;
    typedef typename FUN::DIMSY DIMSY;
    typedef typename FUN::DIMSP DIMSP;
    const int DIMX = DIMSX::SUM;
    const int DIMY = DIMSY::SUM;
    const int DIMP = DIMSP::SUM;
    const int DIMOUT = FUN::DIM; // dimension of output variable
    const int DIMFOUT = DIMSX::FIRST;     // DIMFOUT is dimension of output variable of inner function
    const int SIZEI = DIMSX::SIZE;
    const int SIZEJ = DIMSY::SIZE;
    const int SIZEP = DIMSP::SIZE;

    // pointers to device data
    TYPE *x_d, *y_d, *param_d;

    // device arrays of pointers to device data
    TYPE **px_d, **py_d, **pp_d;

    // single hipMalloc
    void **p_data;
    CudaSafeCall(hipMalloc((void **) &p_data,
                            sizeof(TYPE *) * (SIZEI + SIZEJ + SIZEP)
                                + sizeof(TYPE) * (DIMP + nx * (DIMX - DIMFOUT + DIMOUT) + ny * DIMY)));

    TYPE **p_data_a = (TYPE **) p_data;
    px_d = p_data_a;
    p_data_a += SIZEI;
    py_d = p_data_a;
    p_data_a += SIZEJ;
    pp_d = p_data_a;
    p_data_a += SIZEP;
    TYPE *p_data_b = (TYPE *) p_data_a;
    param_d = p_data_b;
    p_data_b += DIMP;
    x_d = p_data_b;
    p_data_b += nx * (DIMX - DIMFOUT + DIMOUT);
    y_d = p_data_b;

    // host arrays of pointers to device data
    TYPE *phx_d[SIZEI];
    TYPE *phy_d[SIZEJ];
    TYPE *php_d[SIZEP];

    int nvals;
    // if DIMSP is empty (i.e. no parameter), nvals = -1 which could result in a segfault
    if (SIZEP > 0) {
      php_d[0] = param_d;
      nvals = DIMSP::VAL(0);
      CudaSafeCall(hipMemcpy(php_d[0], pp_h[0], sizeof(TYPE) * nvals, hipMemcpyHostToDevice));

      for (int k = 1; k < SIZEP; k++) {
        php_d[k] = php_d[k - 1] + nvals;
        nvals = DIMSP::VAL(k);
        CudaSafeCall(hipMemcpy(php_d[k], pp_h[k], sizeof(TYPE) * nvals, hipMemcpyHostToDevice));
      }
    }

    phx_d[0] = x_d;
    nvals = nx * DIMOUT;
    for (int k = 1; k < SIZEI; k++) {
      phx_d[k] = phx_d[k - 1] + nvals;
      nvals = nx * DIMSX::VAL(k);
      CudaSafeCall(hipMemcpy(phx_d[k], px_h[k], sizeof(TYPE) * nvals, hipMemcpyHostToDevice));
    }

    // if DIMSY is empty (i.e. no Vj variable), nvals = -1 which could result in a segfault
    if (SIZEJ > 0) {
      phy_d[0] = y_d;
      nvals = ny * DIMSY::VAL(0);
      CudaSafeCall(hipMemcpy(phy_d[0], py_h[0], sizeof(TYPE) * nvals, hipMemcpyHostToDevice));

      for (int k = 1; k < SIZEJ; k++) {
        phy_d[k] = phy_d[k - 1] + nvals;
        nvals = ny * (int) DIMSY::VAL(k);
        CudaSafeCall(hipMemcpy(phy_d[k], py_h[k], sizeof(TYPE) * nvals, hipMemcpyHostToDevice));
      }
    }

    // copy arrays of pointers
    CudaSafeCall(hipMemcpy(pp_d, php_d, SIZEP * sizeof(TYPE *), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(px_d, phx_d, SIZEI * sizeof(TYPE *), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(py_d, phy_d, SIZEJ * sizeof(TYPE *), hipMemcpyHostToDevice));

    // Compute on device : grid and block are both 1d
    int dev = -1;
    CudaSafeCall(hipGetDevice(&dev));

    dim3 blockSize;

    SetGpuProps(dev);

    // warning : blockSize.x was previously set to CUDA_BLOCK_SIZE; currently CUDA_BLOCK_SIZE value is used as a bound.
    blockSize.x = ::std::min(CUDA_BLOCK_SIZE,
                             ::std::min(maxThreadsPerBlock, (int) (sharedMemPerBlock / ::std::max(1, (int) (DIMY * sizeof(TYPE)))))); // number of threads in each block

    dim3 gridSize;
    gridSize.x = nx / blockSize.x + (nx % blockSize.x == 0 ? 0 : 1);

    // Size of the SharedData : blockSize.x*(DIMY)*sizeof(TYPE)
    GpuConv1DOnDevice<TYPE> << < gridSize, blockSize, blockSize.x * (DIMY) * sizeof(TYPE) >>
        > (fun, nx, ny, px_d, py_d, pp_d);

    // block until the device has completed
    CudaSafeCall(hipDeviceSynchronize());
    CudaCheckError();

    // Send data from device to host.
    CudaSafeCall(hipMemcpy(*px_h, x_d, sizeof(TYPE) * (nx * DIMOUT), hipMemcpyDeviceToHost));

    // Free memory.
    CudaSafeCall(hipFree(p_data));

    return 0;
  }

// and use getlist to enroll them into "pointers arrays" px and py.
  template<typename TYPE, class FUN, typename... Args>
  static int Eval(FUN fun, int nx, int ny, int device_id, TYPE *x1_h, Args... args) {

    if (device_id != -1)
      CudaSafeCall(hipSetDevice(device_id));

    typedef typename FUN::VARSI VARSI;
    typedef typename FUN::VARSJ VARSJ;
    typedef typename FUN::VARSP VARSP;

    const int SIZEI = VARSI::SIZE + 1;
    const int SIZEJ = VARSJ::SIZE;
    const int SIZEP = VARSP::SIZE;

    using DIMSX = GetDims<VARSI>;
    using DIMSY = GetDims<VARSJ>;
    using DIMSP = GetDims<VARSP>;

    using INDSI = GetInds<VARSI>;
    using INDSJ = GetInds<VARSJ>;
    using INDSP = GetInds<VARSP>;

    TYPE *px_h[SIZEI];
    TYPE *py_h[SIZEJ];
    TYPE *pp_h[SIZEP];

    px_h[0] = x1_h;
    getlist<INDSI>(px_h + 1, args...);
    getlist<INDSJ>(py_h, args...);
    getlist<INDSP>(pp_h, args...);

    return Eval_(fun, nx, ny, px_h, py_h, pp_h);

  }

// same without the device_id argument
  template<typename TYPE, class FUN, typename... Args>
  static int Eval(FUN fun, int nx, int ny, TYPE *x1_h, Args... args) {
    return Eval(fun, nx, ny, -1, x1_h, args...);
  }

// Idem, but with args given as an array of arrays, instead of an explicit list of arrays
  template<typename TYPE, class FUN>
  static int Eval(FUN fun, int nx, int ny, TYPE *x1_h, TYPE **args, int device_id = -1) {

    // We set the GPU device on which computations will be performed
    if (device_id != -1)
      CudaSafeCall(hipSetDevice(device_id));

    typedef typename FUN::VARSI VARSI;
    typedef typename FUN::VARSJ VARSJ;
    typedef typename FUN::VARSP VARSP;

    const int SIZEI = VARSI::SIZE + 1;
    const int SIZEJ = VARSJ::SIZE;
    const int SIZEP = VARSP::SIZE;

    using DIMSX = GetDims<VARSI>;
    using DIMSY = GetDims<VARSJ>;
    using DIMSP = GetDims<VARSP>;

    using INDSI = GetInds<VARSI>;
    using INDSJ = GetInds<VARSJ>;
    using INDSP = GetInds<VARSP>;

    TYPE *px_h[SIZEI];
    TYPE *py_h[SIZEJ];
    TYPE *pp_h[SIZEP];

    px_h[0] = x1_h;
    for (int i = 1; i < SIZEI; i++)
      px_h[i] = args[INDSI::VAL(i - 1)];
    for (int i = 0; i < SIZEJ; i++)
      py_h[i] = args[INDSJ::VAL(i)];
    for (int i = 0; i < SIZEP; i++)
      pp_h[i] = args[INDSP::VAL(i)];

    return Eval_(fun, nx, ny, px_h, py_h, pp_h);

  }

};

struct GpuConv1D_FromDevice {
  template<typename TYPE, class FUN>
  static int Eval_(FUN fun, int nx, int ny, TYPE **phx_d, TYPE **phy_d, TYPE **php_d) {

    typedef typename FUN::DIMSX DIMSX;
    typedef typename FUN::DIMSY DIMSY;
    typedef typename FUN::DIMSP DIMSP;
    const int DIMY = DIMSY::SUM;
    const int SIZEI = DIMSX::SIZE;
    const int SIZEJ = DIMSY::SIZE;
    const int SIZEP = DIMSP::SIZE;

    // device arrays of pointers to device data
    TYPE **px_d, **py_d, **pp_d;

    // single hipMalloc
    void **p_data;
    CudaSafeCall(hipMalloc((void **) &p_data, sizeof(TYPE *) * (SIZEI + SIZEJ + SIZEP)));

    TYPE **p_data_a = (TYPE **) p_data;
    px_d = p_data_a;
    p_data_a += SIZEI;
    py_d = p_data_a;
    p_data_a += SIZEJ;
    pp_d = p_data_a;

    CudaSafeCall(hipMemcpy(px_d, phx_d, SIZEI * sizeof(TYPE *), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(py_d, phy_d, SIZEJ * sizeof(TYPE *), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(pp_d, php_d, SIZEP * sizeof(TYPE *), hipMemcpyHostToDevice));

    // Compute on device : grid and block are both 1d

    int dev = -1;
    CudaSafeCall(hipGetDevice(&dev));

    SetGpuProps(dev);

    dim3 blockSize;
    // warning : blockSize.x was previously set to CUDA_BLOCK_SIZE; currently CUDA_BLOCK_SIZE value is used as a bound.
    blockSize.x = ::std::min(CUDA_BLOCK_SIZE,
                             ::std::min(maxThreadsPerBlock,
                                        (int) (sharedMemPerBlock / ::std::max(1,
                                                                              (int) (DIMY
                                                                                  * sizeof(TYPE)))))); // number of threads in each block

    dim3 gridSize;
    gridSize.x = nx / blockSize.x + (nx % blockSize.x == 0 ? 0 : 1);

    // Size of the SharedData : blockSize.x*(DIMY)*sizeof(TYPE)
    GpuConv1DOnDevice<TYPE> << < gridSize, blockSize, blockSize.x * (DIMY) * sizeof(TYPE) >>
        > (fun, nx, ny, px_d, py_d, pp_d);

    // block until the device has completed
    CudaSafeCall(hipDeviceSynchronize());
    CudaCheckError();

    CudaSafeCall(hipFree(p_data));

    return 0;
  }

// Same wrappers, but for data located on the device
  template<typename TYPE, class FUN, typename... Args>
  static int Eval(FUN fun, int nx, int ny, int device_id, TYPE *x1_d, Args... args) {

    // device_id is provided, so we set the GPU device accordingly
    // Warning : is has to be consistent with location of data
    CudaSafeCall(hipSetDevice(device_id));

    typedef typename FUN::VARSI VARSI;
    typedef typename FUN::VARSJ VARSJ;
    typedef typename FUN::VARSP VARSP;

    const int SIZEI = VARSI::SIZE + 1;
    const int SIZEJ = VARSJ::SIZE;
    const int SIZEP = VARSP::SIZE;

    using DIMSX = GetDims<VARSI>;
    using DIMSY = GetDims<VARSJ>;
    using DIMSP = GetDims<VARSP>;

    using INDSI = GetInds<VARSI>;
    using INDSJ = GetInds<VARSJ>;
    using INDSP = GetInds<VARSP>;

    TYPE *phx_d[SIZEI];
    TYPE *phy_d[SIZEJ];
    TYPE *php_d[SIZEP];

    phx_d[0] = x1_d;

    getlist<INDSI>(phx_d + 1, args...);
    getlist<INDSJ>(phy_d, args...);
    getlist<INDSP>(php_d, args...);

    return Eval_(fun, nx, ny, phx_d, phy_d, php_d);

  }

// same without the device_id argument
  template<typename TYPE, class FUN, typename... Args>
  static int Eval(FUN fun, int nx, int ny, TYPE *x1_d, Args... args) {
    // We set the GPU device on which computations will be performed
    // to be the GPU on which data is located.
    // NB. we only check location of x1_d which is the output vector
    // so we assume that input data is on the same GPU
    // note : hipPointerGetAttributes has a strange behaviour:
    // it looks like it makes a copy of the vector on the default GPU device (0) !!! 
    // So we prefer to avoid this and provide directly the device_id as input (first function above)
    hipPointerAttribute_t attributes;
    CudaSafeCall(hipPointerGetAttributes(&attributes, x1_d));
    return Eval(fun, nx, ny, attributes.device, x1_d, args...);
  }

  template<typename TYPE, class FUN>
  static int Eval(FUN fun, int nx, int ny, TYPE *x1_d, TYPE **args, int device_id = -1) {

    if (device_id == -1) {
      // We set the GPU device on which computations will be performed
      // to be the GPU on which data is located.
      // NB. we only check location of x1_d which is the output vector
      // so we assume that input data is on the same GPU
      // note : hipPointerGetAttributes has a strange behaviour:
      // it looks like it makes a copy of the vector on the default GPU device (0) !!!
      // So we prefer to avoid this and provide directly the device_id as input (else statement below)
      hipPointerAttribute_t attributes;
      CudaSafeCall(hipPointerGetAttributes(&attributes, x1_d));
      CudaSafeCall(hipSetDevice(attributes.device));
    } else // device_id is provided, so we use it. Warning : is has to be consistent with location of data
      CudaSafeCall(hipSetDevice(device_id));

    typedef typename FUN::VARSI VARSI;
    typedef typename FUN::VARSJ VARSJ;
    typedef typename FUN::VARSP VARSP;

    const int SIZEI = VARSI::SIZE + 1;
    const int SIZEJ = VARSJ::SIZE;
    const int SIZEP = VARSP::SIZE;

    using DIMSX = GetDims<VARSI>;
    using DIMSY = GetDims<VARSJ>;
    using DIMSP = GetDims<VARSP>;

    using INDSI = GetInds<VARSI>;
    using INDSJ = GetInds<VARSJ>;
    using INDSP = GetInds<VARSP>;

    TYPE *px_d[SIZEI];
    TYPE *py_d[SIZEJ];
    TYPE *pp_d[SIZEP];

    px_d[0] = x1_d;
    for (int i = 1; i < SIZEI; i++)
      px_d[i] = args[INDSI::VAL(i - 1)];
    for (int i = 0; i < SIZEJ; i++)
      py_d[i] = args[INDSJ::VAL(i)];
    for (int i = 0; i < SIZEP; i++)
      pp_d[i] = args[INDSP::VAL(i)];

    return Eval_(fun, nx, ny, px_d, py_d, pp_d);

  }

};

}
